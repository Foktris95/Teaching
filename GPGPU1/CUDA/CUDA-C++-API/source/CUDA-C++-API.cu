#include "hip/hip_runtime.h"
#include <CUDA-C++-API.hpp>


int main(void)
{
  int N = 1<<20;
  std::vector<float> x(N, 1.0f), y(N, 2.0f);
  float *d_x = nullptr, *d_y = nullptr, a = 2.0;

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  hipMemcpy(d_x, x.data(), N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), N*sizeof(float), hipMemcpyHostToDevice);

  auto saxpy = [=] __device__ ()
  {
      int i = blockIdx.x*blockDim.x + threadIdx.x;
      
      if (i < N) d_y[i] = a * d_x[i] + d_y[i];
  };

  // Perform SAXPY on 1M elements
  cuda::launch_kernel<<<(N+255)/256, 256>>>(saxpy);

  hipMemcpy(y.data(), d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float res = a * 1.0f + 2.0f;
  float maxError = *std::max_element(y.cbegin(),
                                     y.cend(),
                                     [=](const float& lhs, const float& rhs)
  {
    return std::abs(lhs - res) < std::abs(rhs - res);
  });

  std::cout.precision(16);
  std::cout << "Max error: " << std::abs(maxError - res) << std::endl;

  hipFree(d_x);
  hipFree(d_y);
}
